#include "hip/hip_runtime.h"
#include "cuMedianFilter.cuh"

/* Insertion sort */
__device__ void d_insertionSort(float* arr, int size)
{
	float curr = 0;
	int j = 0;
	for (int i = 1; i < size; i++)
	{
		curr = arr[i];
		j = i - 1;

		while (j >= 0 && arr[j] > curr)
		{
			arr[j + 1] = arr[j];
			j = j - 1;
		}
		arr[j + 1] = curr;
	}
}

/* Median Filter */
__device__ int borderReplicate(int idx, int size)
{
	if (idx < 0)
	{
		return 0;
	}
	else if (idx >= size)
	{
		return size - 1;
	}
	else
	{
		return idx;
	}
}

__device__ float d_borderInterpolate(float* d_inputImage, int x, int y, int width, int height, int borderType)
{
	switch (borderType)
	{
	case CUDA_BORDER_REPLICATE:
		x = borderReplicate(x, width);
		y = borderReplicate(y, height);
	}
	return d_inputImage[x + y * width];
}

__global__ void d_medianBlur(float* d_inputImage, float* d_outputImage, int width, int height, int kernel_x, int kernel_y, int borderType, bool finiteOnly)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;	// Width
	int y = blockIdx.y * blockDim.y + threadIdx.y;	// Height

	if (x < width && y < height)
	{
		int kw05 = kernel_x >> 1;
		int kh05 = kernel_y >> 1;
		int cnt = 0;
		float kernel[256] = { 0 };	// CHK: Dynamic memory allocation ���� �ʿ�
		float curr_val = 0;
		for (int j = -kh05; j <= kh05; j++)
		{
			for (int i = -kw05; i <= kw05; i++)
			{
				if (x - kw05 >= 0 && y - kh05 >= 0 && x + kw05 < width && y + kh05 < height)
				{
					curr_val = d_inputImage[(y + j) * width + (x + i)];
				}
				else
				{
					curr_val = d_borderInterpolate(d_inputImage, x + i, y + j, width, height, borderType); // CHK: Border Type �پ�ȭ �ʿ�
				}

				if (finiteOnly)
				{
					if (isfinite(curr_val))
					{
						kernel[cnt] = curr_val;
						cnt++;
					}
				}
				else
				{
					kernel[cnt] = curr_val;
					cnt++;
				}
			}
		}

		int threshold = 0;
		if (finiteOnly)
		{
			threshold = (kernel_x * kernel_y) / 2;
		}

		if (cnt > threshold)
		{
			d_insertionSort(kernel, cnt);
			d_outputImage[width * y + x] = kernel[cnt >> 1];
		}
		else
		{
			d_outputImage[width * y + x] = d_inputImage[width * y + x];
		}
	}
}

void cuMedianBlur(cv::Mat h_inputImage, cv::Mat& h_outputImage, int kernel_x, int kernel_y, int borderType, bool finiteOnly)
{
	//TIMER_CUDA();
	if (h_inputImage.empty() || h_inputImage.channels() != 1)
	{
		h_outputImage = h_inputImage.clone();
		return;
	}

	int src_type = h_inputImage.type();
	h_inputImage.convertTo(h_inputImage, CV_32F);

	float* d_inputImage, * d_outputImage;
	int size = h_inputImage.rows * h_inputImage.cols * sizeof(float);

	/* Memory Allocation */
	CUDA_CHECK(hipMalloc(&d_inputImage, size));
	CUDA_CHECK(hipMalloc(&d_outputImage, size));

	hipStream_t stream;
	CUDA_CHECK(hipStreamCreate(&stream));

	/* Memory Copy (Host to Device) */
	CUDA_CHECK(hipMemcpyAsync(d_inputImage, (float*)h_inputImage.data, size, hipMemcpyHostToDevice, stream));

	/* Run Kernel Function */
	dim3 dimBlock(8, 8);	// (width, height) (��� �� �ִ� ������ ��: 1024)
	dim3 dimGrid(ceil(h_inputImage.cols / (double)dimBlock.x), ceil(h_inputImage.rows / (double)dimBlock.y));	// (width, height)
	d_medianBlur <<< dimGrid, dimBlock, 0, stream >>> (d_inputImage, d_outputImage, h_inputImage.cols, h_inputImage.rows, kernel_x, kernel_y, borderType, finiteOnly);

	/* Memory Copy (Device to Host) */
	CUDA_CHECK(hipMemcpyAsync((float*)h_outputImage.data, d_outputImage, size, hipMemcpyDeviceToHost, stream));

	h_outputImage.convertTo(h_outputImage, src_type);

	/* Memory Deallocation */
	CUDA_CHECK(hipStreamDestroy(stream));
	CUDA_CHECK(hipFree(d_inputImage));
	CUDA_CHECK(hipFree(d_outputImage));
}